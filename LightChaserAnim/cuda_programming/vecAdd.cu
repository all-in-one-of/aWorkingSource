
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel definition
__global__
void VecAdd(float* A, float* B, float* C,int* msg)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
    *msg = i;
}

int main()
{
	int N = 256;
	float *A = new float[N];
	float *B = new float[N];
	float *C = new float[N];
	int *msg;
	hipMallocManaged(&A, N*sizeof(float));
	hipMallocManaged(&B, N*sizeof(float));
	hipMallocManaged(&C, N*sizeof(float));
	hipMallocManaged(&msg, sizeof(int));
	for(int i=0;i<N;i++)
	{
		A[i] = 0.1;
		B[i] = 0.1;
	}

	// Kernel invocation with N threads
	int blockSize = 256;
	VecAdd<<<1, blockSize>>>(A, B, C, msg);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	std::cout << *msg << std::endl;

	hipFree(A);
	hipFree(B);
	hipFree(C);
	hipFree(msg);
}
